#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <iostream>

int main() {
    float *d_input, *d_kernel, *d_output_custom, *d_output_cudnn;
    hipMalloc(&d_input, B*C*H*W*sizeof(float));
    hipMalloc(&d_kernel, K*C*KH*KW*sizeof(float));
    hipMalloc(&d_output_custom, B*K*H*W*sizeof(float));
    hipMalloc(&d_output_cudnn, B*K*H*W*sizeof(float));

    // Copy input and kernel to device
    hipMemcpy(d_input, h_input, B*C*H*W*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel, K*C*KH*KW*sizeof(float), hipMemcpyHostToDevice);

    

    // Kernel launch configuration
    dim3 blockDim(H, W);
    dim3 gridDim(B, K);
    custom_conv2d_kernel<<<gridDim, blockDim>>>(d_input, d_output_custom, d_kernel, B, C, H, W, K, KH, KW);

    // Copy custom kernel output back to host
    hipMemcpy(h_output_custom, d_output_custom, B*K*H*W*sizeof(float), hipMemcpyDeviceToHost);

    // cuDNN setup
    hipdnnHandle_t cudnn;
		  hipdnnCreate(&cudnn);
		
		hipdnnTensorDescriptor_t inputDesc, outputDesc;
		hipdnnFilterDescriptor_t filterDesc;
		hipdnnConvolutionDescriptor_t convDesc;
		hipdnnCreateTensorDescriptor(&inputDesc);
    hipdnnCreateTensorDescriptor(&outputDesc);
    hipdnnCreateFilterDescriptor(&filterDesc);
    hipdnnCreateConvolutionDescriptor(&convDesc);

    hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, B, C, H, W);
    hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, K, C, KH, KW);
    hipdnnSetConvolution2dDescriptor(convDesc, 0, 0, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT);

    int n, c, h, w;
    hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc, &n, &c, &h, &w);
    hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w);

    hipdnnConvolutionFwdAlgo_t algo;
    hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc, &n, &c, &h, &w);

    size_t workspaceSize;
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn, inputDesc, filterDesc, convDesc, outputDesc, algo, &workspaceSize);

    void* workspace;
    hipMalloc(&workspace, workspaceSize);

    const float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn, &alpha, inputDesc, d_input, filterDesc, d_kernel, convDesc, algo, workspace, workspaceSize, &beta, outputDesc, d_output_cudnn));

}